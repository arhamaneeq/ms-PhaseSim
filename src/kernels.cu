#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void markovSweep(Cell* d_input, int w, int h, float T, float mu, hiprandState* states, int offset, float J) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * w + x;
    if (x >= w || y >= h) return;
    if ((x + y) % 2 != offset) return;

    hiprandState localState = states[idx];
    float r1 = hiprand_uniform(&localState);
    float r2 = hiprand_uniform(&localState);
    states[idx] = localState;

    int delN;
    // if (d_input[idx] == 0) {
    //     delN = 1;
    // } else if (d_input[idx] == 255) {
    //     delN = -1;
    // } else {
    //     delN = (r1 < 0.5f) ? -1 : +1;
    // }

    delN = (r1 <= 0.5) ? -1 : +1;

    float delE = deltaE(d_input, w, h, x, y, delN, J);
    // float k = 1;
    // float beta = 1 / (k * T);
    float delPhi = delE - mu * delN;
    float p_acc;

    if (T <= 1e-6f) { p_acc = fminf(1.0f, expf(-delPhi / 1e-6f)); }
    else { p_acc = fminf(1.0f, expf(-delPhi / T)); }

    if (r2 < p_acc) {
        d_input[idx] = (delN == 1) ? true : false;
    }
}

__global__ void initRNG(hiprandState* states, unsigned long seed, int w) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * w + x;

    hiprand_init(seed, idx, 0, &states[idx]);
}

__device__ float deltaE(const Cell* d_input, int w, int h, int x, int y, int delN, float J) {
    int xL = (x == 0) ? w - 1 : x - 1;
    int xR = (x == w - 1) ? 0 : x + 1;
    int yU = (y == 0) ? h - 1 : y - 1;
    int yD = (y == h - 1) ? 0 : y + 1;
    
    float sumN  = occupancy(d_input[y * w + xL])
                + occupancy(d_input[y * w + xR])
                + occupancy(d_input[yU * w + x])
                + occupancy(d_input[yD * w + x]);


    float delS = delN;
    float deltaE = - J * (float)  delS * sumN;
    return deltaE;

}

__device__ float occupancy(Cell v) {
    return v ? 1.0f : 0.0f;
}