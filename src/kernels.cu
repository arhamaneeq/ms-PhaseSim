#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void markovSweep(uint8_t* d_input, int w, int h, float T, float mu, hiprandState* states, int offset) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * w + x;
    if (x >= w || y >= h) return;
    if ((x + y) % 2 != offset) return;

    hiprandState localState = states[idx];
    float r1 = hiprand_uniform(&localState);
    float r2 = hiprand_uniform(&localState);
    states[idx] = localState;

    int delN;
    if (d_input[idx] == 0) {
        delN = -1;
    } else if (d_input[idx] == 255) {
        delN = +1;
    } else {
        delN = (r1 < 0.5f) ? -1 : +1;
    }

    float J = 1.0;
    float delE = deltaE(d_input, w, h, x, y, delN, J, 0);                 // TODO: define hamilltonian
    // float k = 1;                    // TODO: remove units, move to dimensionsless everythingggg
    // float beta = 1 / (k * T);
    float delPhi = delE - mu * delN;

    float p_acc = fminf(1.0f , expf(-delPhi / T));


    if (r2 < p_acc) {
        d_input[idx] += delN;
    }
}

__global__ void initRNG(hiprandState* states, unsigned long seed, int w) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * w + x;

    hiprand_init(seed, idx, 0, &states[idx]);
}

__device__ float deltaE(const uint8_t* d_input, int w, int h, int x, int y, int delN, float J, float eps) {
    int xL = (x == 0) ? w - 1 : x - 1;
    int xR = (x == w - 1) ? 0 : x + 1;
    int yU = (y == 0) ? h - 1 : y - 1;
    int yD = (y == h - 1) ? 0 : y + 1;
    
    float sumN  = spinVal(d_input[y * w + xL])
                + spinVal(d_input[y * w + xR])
                + spinVal(d_input[yU * w + x])
                + spinVal(d_input[yD * w + x]);

    float deltaE = - J * (float)  delN * sumN + eps * (float) delN;
    return deltaE;

}

__device__ float spinVal(uint8_t v) {
    return (v > 127) ? 1.0f : -1.0f;
}