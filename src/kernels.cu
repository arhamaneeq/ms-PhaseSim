#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__ void markovStep(uint8_t* d_input, int w, int h, float T, float mu, hiprandState* states, int offset) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int idx = y * w + x;
    if (x >= w || y >= h) return;
    if ((x + y) % 2 != offset) return;

    hiprandState localState = states[idx];
    float r1 = hiprand_uniform(&localState);
    float r2 = hiprand_uniform(&localState);
    states[idx] = localState;

    int delN;
    if (d_input[idx] == 0) {
        delN = +1;
    } else if (d_input[idx] == 255) {
        delN = -1;
    } else {
        delN = (r1 < 0.5f) ? -1 : +1;
    }

    float J = 1.0;
    float delE = deltaE(d_input, w, h, x, y, delN, J, 0);                 // TODO: define hamilltonian
    // float k = 1;                    // TODO: remove units, move to dimensionsless everythingggg
    // float beta = 1 / (k * T);
    float delPhi = delE - mu * delN;

    float p_acc = fminf(1.0f , expf(-delPhi / T));


    if (r2 < p_acc) {
        d_input[idx] += delN;
    }
}

__global__ void initRNG(hiprandState* states, unsigned long seed) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    hiprand_init(seed, idx, 0, &states[idx]);
}

__device__ float deltaE(const uint8_t* d_input, int w, int h, int x, int y, int delN, float J, float eps) {
    int idx = y * w + x;

    int xL = (x == 0) ? w - 1 : x - 1;
    int xR = (x == w - 1) ? 0 : x + 1;
    int yU = (y == 0) ? h - 1 : y - 1;
    int yD = (y == h - 1) ? 0 : y + 1;
    
    float sumN  = (float) d_input[y * w + xL]
                + (float) d_input[y * w + xR]
                + (float) d_input[yU * w + x]
                + (float) d_input[yD * w + x];

    float deltaE = - J * (float)  delN * sumN + eps * (float) delN;
    return deltaE;

}