#include "wrapper.cuh"

void* allocateDeviceMemory(size_t bytes) {
    void* ptr = nullptr;
    hipError_t err = hipMalloc(&ptr, bytes);

    if (err!=hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return ptr;
}

void deallocateDeviceMemory(void* ptr) {
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}

void copyMemory(void* dst, const void* src, size_t bytes, int direction) {
    hipError_t err = hipMemcpy(dst, src, bytes, static_cast<hipMemcpyKind>(direction));

    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}