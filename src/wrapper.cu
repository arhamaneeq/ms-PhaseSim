#include "wrapper.cuh"

void* allocateDeviceMemory(size_t bytes) {
    void* ptr = nullptr;
    hipError_t err = hipMalloc(&ptr, bytes);

    if (err!=hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }

    return ptr;
}

void deallocateDeviceMemory(void* ptr) {
    hipError_t err = hipFree(ptr);
    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}

void copyMemory(void* dst, const void* src, size_t bytes, int direction) {
    hipError_t err = hipMemcpy(dst, src, bytes, static_cast<hipMemcpyKind>(direction));

    if (err != hipSuccess) {
        throw std::runtime_error(hipGetErrorString(err));
    }
}

void markovStep(uint8_t* d_cells, int w, int h, float T, float mu, hiprandState* states) {
    dim3 block(16, 16);
    dim3 grid((w + block.x - 1) / block.x,
              (h + block.y - 1) / block.y
    );

    markovSweep<<<grid, block>>>(d_cells, w, h, T, mu, states, 0);
    markovSweep<<<grid, block>>>(d_cells, w, h, T, mu, states, 1);

    hipDeviceSynchronize();
}

hiprandState* genRands(int w, int h) {
    hiprandState* d_states;
    hipMalloc(&d_states, w * h * sizeof(hiprandState));

    dim3 block(16, 16);
    dim3 grid((w + block.x - 1) / block.x,
              (h + block.y - 1) / block.y);

    initRNG<<<grid, block>>>(d_states, 42);
    hipDeviceSynchronize();

    return d_states;
}